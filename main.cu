#include "xeon_plasma.h"
#include <stdlib.h>
#include "mpi_shortcut.h"

int main(int argc,char*argv[])
{
      XeonPlasma<GPUCell> *plasma;
   
      InitMPI(argc,argv);

      size_t sizeP;

      printf("oarticle size %d %d \n",sizeof(Particle),sizeof(Particle)/sizeof(double));
//      hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);

//      printf("printf default limit %d \n",sizeP/1024/1024);

      sizeP *= 10;
      sizeP *= 10;
      sizeP *= 10;
      sizeP *= 10;
//      hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeP);

//      hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);



//      printf("printf limit set to %d \n",sizeP/1024/1024);

   
      int err = SetDevice(0);
   
//      printf("err %d \n",err);

   //plasma = new GPUPlasma<GPUCell>(100,4,4,1.2566,0.05,0.05,1.0,100,1.0,0.001);
   plasma = new XeonPlasma<GPUCell>(100,4,4,1.1424,0.05,0.05,1.0,2000,1.0,0.001);
 
   plasma->Initialize();


   

   double t = plasma->compareCPUtoGPU();
   printf("----------------------------------------------------------- plasma check before move %.5f\n",t);
   size_t m_free,m_total;

   GetDeviceMemory(&m_free,&m_total);
   struct sysinfo info;


   for(int nt = START_STEP_NUMBER;nt <= TOTAL_STEPS;nt++)
   {
	   GetDeviceMemory(&m_free,&m_total);
	   sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("before Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024,m_total/1024/1024,m_free/1024/1024);
#endif

       plasma->Step(nt);

       GetDeviceMemory(&m_free,&m_total);
       sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("after  Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024/1024,m_total/1024/1024/1024,m_free/1024/1024/1024);
#endif
   }

   t = plasma->compareCPUtoGPU();
   printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ plasma check after move %.5f\n",t);

   delete plasma;
   
   CloseMPI();

   return 0;
}
